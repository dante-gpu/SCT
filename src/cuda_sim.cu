#include "hip/hip_runtime.h"
#include "cuda_sim.h"
#include <iostream>
#include <vector>
#include <tuple>
#include <cmath>

__global__ void calculate_cosine_similarity_kernel(
    const float* vec_i, const float* vec_j, float* result, int vec_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= vec_size) return;

    float dot_product = 0.0f;
    float norm_i = 0.0f;
    float norm_j = 0.0f;

    for (int k = 0; k < vec_size; ++k) {
        dot_product += vec_i[k] * vec_j[k];
        norm_i += vec_i[k] * vec_i[k];
        norm_j += vec_j[k] * vec_j[k];
    }

    norm_i = sqrtf(norm_i);
    norm_j = sqrtf(norm_j);

    if (norm_i == 0.0f || norm_j == 0.0f) {
        *result = 0.0f;
    } else {
        *result = dot_product / (norm_i * norm_j);
    }
}

std::vector<std::tuple<int, int, float>> calculate_similarity_cuda(
    const std::vector<std::vector<float>>& tf_vectors,
    float threshold) {
    std::vector<std::tuple<int, int, float>> results;
    int num_docs = tf_vectors.size();

    for (int i = 0; i < num_docs; ++i) {
        for (int j = i + 1; j < num_docs; ++j) {
            int vec_size = tf_vectors[i].size();

            float* dev_vec_i;
            float* dev_vec_j;
            float* dev_result;

            hipMalloc((void**)&dev_vec_i, vec_size * sizeof(float));
            hipMalloc((void**)&dev_vec_j, vec_size * sizeof(float));
            hipMalloc((void**)&dev_result, sizeof(float));

            hipMemcpy(dev_vec_i, tf_vectors[i].data(), vec_size * sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(dev_vec_j, tf_vectors[j].data(), vec_size * sizeof(float), hipMemcpyHostToDevice);

            calculate_cosine_similarity_kernel<<<1, 1>>>(dev_vec_i, dev_vec_j, dev_result, vec_size);

            float similarity;
            hipMemcpy(&similarity, dev_result, sizeof(float), hipMemcpyDeviceToHost);

            hipFree(dev_vec_i);
            hipFree(dev_vec_j);
            hipFree(dev_result);

            if (similarity >= threshold) {
                results.emplace_back(i, j, similarity);
            }
        }
    }

    return results;
}
